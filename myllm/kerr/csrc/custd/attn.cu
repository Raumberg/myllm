#include "hip/hip_runtime.h"
// taken from https://github.com/vllm-project/vllm/blob/main/csrc/attention/attention_utils.cuh

// Q*K^T operation.
template <int THREAD_GROUP_SIZE, typename Vec, int N>
inline __device__ float qk_dot_(const Vec (&q)[N], const Vec (&k)[N]) {
  using A_vec = typename FloatVec<Vec>::Type;
  // Compute the parallel products for Q*K^T (treat vector lanes separately).
  A_vec qk_vec = mul<A_vec, Vec, Vec>(q[0], k[0]);
#pragma unroll
  for (int ii = 1; ii < N; ++ii) {
    qk_vec = vllm::fma(q[ii], k[ii], qk_vec);
  }

  // Finalize the reduction across lanes.
  float qk = sum(qk_vec);
#pragma unroll
  for (int mask = THREAD_GROUP_SIZE / 2; mask >= 1; mask /= 2) {
    qk += VLLM_SHFL_XOR_SYNC(qk, mask);
  }
  return qk;
}

template <typename T, int THREAD_GROUP_SIZE>
struct Qk_dot {
  template <typename Vec, int N>
  static inline __device__ float dot(const Vec (&q)[N], const Vec (&k)[N]) {
    return qk_dot_<THREAD_GROUP_SIZE>(q, k);
  }
};